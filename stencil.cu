
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void stencil(float* in, float* out, int N, int n, int BLOCKSIZE){
	if (blockIdx.x==0){
		__shared__ float shared_in[22];//BLOCKSIZE+2*n];
		int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
		shared_in[threadIdx.x+n] = in[globalIdx];
		if (threadIdx.x>(BLOCKSIZE-n-1)){
			shared_in[threadIdx.x+2*n] = in[globalIdx+n];
		}
		__syncthreads();
		int value = 0;
		if (threadIdx.x>n-1){
			for (int i=0; i<2*n+1; i++){
				value += shared_in[threadIdx.x+i];
			}
		}
		out[globalIdx] = value;
		
	}
	else if (blockIdx.x==(int(N/BLOCKSIZE)-1)){
		__shared__ float shared_in[22];
		int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
		shared_in[threadIdx.x+n] = in[globalIdx];
		if (threadIdx.x<n){
			shared_in[threadIdx.x] = in[globalIdx-n];
		}
		__syncthreads();
		int value = 0;
		if (threadIdx.x<BLOCKSIZE-n){
			for (int i=0; i<2*n+1; i++){
				value += shared_in[threadIdx.x+i];
			}
		}
		out[globalIdx] = value;
	}
	else{
		__shared__ float shared_in[22];
		int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
		shared_in[threadIdx.x+n] = in[globalIdx];
		if (threadIdx.x<n){
			shared_in[threadIdx.x] = in[globalIdx-n];
		}
		if (threadIdx.x>BLOCKSIZE-n-1){
			shared_in[threadIdx.x+2*n] = in[globalIdx+n];
		}
		__syncthreads();
		int value = 0;
		for (int i=0; i<2*n+1; i++){
			value += shared_in[threadIdx.x+i];
		}
		out[globalIdx] = value;
	}
}


int main(){
	float* h_a = NULL;
	float* h_b = NULL;
	float* d_a = NULL;
	float* d_b = NULL;

	int BLOCKSIZE = 16;
	int N = 512;
	int n = 3;

	h_a = (float*)malloc(N*sizeof(float));
	h_b = (float*)malloc(N*sizeof(float));
	hipMalloc((void**)&d_a, N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));

	if ((h_a==NULL)||(d_a==NULL)||(h_b==NULL)&&(d_b==NULL)){
		printf("Cannot allocate memory.\n");
	}

	memset(h_b, 0, N*sizeof(float));
	for (int i=0; i<N; i++){
		h_a[i]=i;
	}

	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);
	
	stencil<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>> (d_a, d_b, N, n, BLOCKSIZE);

	hipMemcpy(h_b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);	
	for (int i=0; i<N; i++){
		printf("A[%d]=%f\n",i,h_b[i]);
	}
	return 0;

}
