
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void stencil(float* in, float* out, int N, int M, int n, int BLOCKSIZE){
	__shared__ float shared_in[22][22];//BLOCKSIZE+2*n;
	int globalIdx_x = blockIdx.x * blockDim.x + threadIdx.x;
	int globalIdx_y = blockIdx.y * blockDim.y + threadIdx.y;
	shared_in[threadIdx.y+n][threadIdx.x+n] = in[globalIdx_y*M + globalIdx_x];
	if (blockIdx.x>0){	
		if (threadIdx.x<n){
			shared_in[threadIdx.y+n][threadIdx.x] = in[(globalIdx_y)*M+globalIdx_x-n];
		}
	}
	if (blockIdx.y>0){	
		if (threadIdx.y<n){
			shared_in[threadIdx.y][threadIdx.x+n] = in[(globalIdx_y-n)*M+globalIdx_x];
		}
	}
	if (blockIdx.x<(int(M/BLOCKSIZE)-1)){
		if (threadIdx.x>BLOCKSIZE-n-1){	
			shared_in[threadIdx.y+n][threadIdx.x+2*n] = in[(globalIdx_y)*M+globalIdx_x+n];
		}
	}
	if (blockIdx.y<(int(N/BLOCKSIZE)-1)){
		if (threadIdx.y>BLOCKSIZE-n-1){
			shared_in[threadIdx.y+2*n][threadIdx.y+n] = in[(globalIdx_y+n)*M+globalIdx_x];
		}
	}
	__syncthreads();
	int value = 0;
	if ((globalIdx_x>=n)&&(globalIdx_x<M-n)&&(globalIdx_y>=n)&&(globalIdx_y<N-n))
	{
		for (int i=0; i<2*n+1; i++){
			value += shared_in[threadIdx.y+i][threadIdx.x+n];
			value += shared_in[threadIdx.y+n][threadIdx.x+i];
		}
		value = value - shared_in[threadIdx.y+n][threadIdx.x+n];	
	}
out[globalIdx_y*M + globalIdx_x] = value;
//	printf("out[%d][%d]=%f\n",globalIdx_y, globalIdx_x, out[globalIdx_y*M+globalIdx_x]);
}


int main(){
	float* h_a = NULL;
	float* h_b = NULL;
	float* d_a = NULL;
	float* d_b = NULL;

	int BLOCKSIZE = 16;
	int N = 512;
	int M = 256;
	int n = 3;

	h_a = (float*)malloc(M*N*sizeof(float));
	h_b = (float*)malloc(M*N*sizeof(float));
	hipMalloc((void**)&d_a, M*N*sizeof(float));
	hipMalloc((void**)&d_b, M*N*sizeof(float));

	if ((h_a==NULL)||(d_a==NULL)||(h_b==NULL)&&(d_b==NULL)){
		printf("Cannot allocate memory.\n");
	}

	memset(h_b, 0, M*N*sizeof(float));
	for (int i=0; i<N; i++){
		for (int j=0; j<M; j++){
			h_a[i*M+j]=i+j;
		}
	}

	hipMemcpy(d_a, h_a, M*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, M*N*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid((M+BLOCKSIZE-1)/BLOCKSIZE, (N+BLOCKSIZE-1)/BLOCKSIZE, 1);
	dim3 block(BLOCKSIZE,BLOCKSIZE,1);	
	stencil<<<grid, block>>> (d_a, d_b, N, M, n, BLOCKSIZE);

	hipMemcpy(h_b, d_b, M*N*sizeof(float), hipMemcpyDeviceToHost);	
	for (int i=0; i<N; i++){
		for (int j=0; j<M; j++){
			printf("A[%d][%d]=%f\n",i,j,h_b[i*M+j]);
		}
	}
	return 0;

}
